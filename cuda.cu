#include "hip/hip_runtime.h"
#include <iostream>
#include "saxpy.h"

__global__ void saxpy(size_t n, real_t a, real_t *x, real_t *y)
{
	const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t stride = blockDim.x * gridDim.x;
	
	for (size_t i = index; i < n; i += stride) {
		y[i] += a * x[i];
	}
}

int main(void)
{
	real_t *x, *y;

	hipMallocManaged(&x, N * sizeof(real_t));
	hipMallocManaged(&y, N * sizeof(real_t));
  
	// initialize x and y arrays on the host
	for (size_t i = 0; i < N; i++) {
		x[i] = XVAL;
		y[i] = YVAL;
	}

	// Run kernel on 1M elements on the CPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	saxpy_timer t;
	
	saxpy<<<numBlocks, blockSize>>>(N, AVAL, x, y);

	hipDeviceSynchronize();
	
	// Check for errors (all values should be 3.0f)
	saxpy_verify(y);
	std::cout << "Total elapsed: " << t.elapsed_msec() << " ms" << std::endl;
	
	// Free memory
	hipFree(x);
	hipFree(y);
	
	return 0;
}
